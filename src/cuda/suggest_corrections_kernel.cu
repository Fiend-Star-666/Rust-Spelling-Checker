
#include <hip/hip_runtime.h>
#define DICTIONARY_SIZE 10000  // Replace 10000 with the size of your dictionary

__device__ char* dictionary[DICTIONARY_SIZE];
__device__ int dictionary_size = DICTIONARY_SIZE;

__device__ size_t cuda_strlen(const char *str) {
    const char *s;
    for (s = str; *s; ++s);
    return s - str;
}

__device__ int wagner_fischer(char* word1, char* word2) {
    int len1 = cuda_strlen(word1);
    int len2 = cuda_strlen(word2);

    // Create two arrays to store the current and previous row of the matrix
    int prev_row[DICTIONARY_SIZE + 1];
    int curr_row[DICTIONARY_SIZE + 1];

    // Initialize the first row of the matrix
    for (int j = 0; j <= len2; j++) {
        prev_row[j] = j;
    }

    // Fill in the rest of the matrix
    for (int i = 1; i <= len1; i++) {
        curr_row[0] = i;
        for (int j = 1; j <= len2; j++) {
            int cost = (word1[i - 1] == word2[j - 1]) ? 0 : 1;
            curr_row[j] = min(min(prev_row[j] + 1, curr_row[j - 1] + 1), prev_row[j - 1] + cost);
        }
        // Swap the current and previous row for the next iteration
        for (int j = 0; j <= len2; j++) {
            prev_row[j] = curr_row[j];
        }
    }

    // The Levenshtein distance is the value in the last cell of the final row
    int result = prev_row[len2];

    return result;
}

extern "C"
__global__ void suggest_corrections_kernel(char** unknown_words, char** corrections, int num_words) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < num_words) {
        char* word = unknown_words[idx];

        // Initialize the minimum distance to a large number
        int min_distance = INT_MAX;

        // For each word in the dictionary
        for (int i = 0; i < dictionary_size; i++) {
            char* dict_word = dictionary[i];

            // Calculate the Wagner-Fischer distance between the unknown word and the dictionary word
            int distance = wagner_fischer(word, dict_word);

            // If the distance is less than the minimum distance, update the minimum distance and the correction
            if (distance < min_distance) {
                min_distance = distance;
                corrections[idx] = dict_word;
            }
        }
    }
}